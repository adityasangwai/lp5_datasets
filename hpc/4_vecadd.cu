
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void add(int* A, int* B, int* C, int size) {

    //calculates the unique thread index (tid) for each thread.
    /*
    blockIdx.x is the block index in the grid.
    blockDim.x is the number of threads in a block.
    threadIdx.x is the index of the current thread within its block*/
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    /*checks whether the calculated thread index (tid) is within the bounds of the vector size (size).
    This ensures that each thread only processes valid elements within the range of the vectors.*/
    if (tid < size) {

        // performs the element-wise addition of vectors A and B at the index tid.
        C[tid] = A[tid] + B[tid];
    }
}

// initializes the A and B vectors with random integers between 0 and 9.
void initialize(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 10;
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main() {
    int N = 8;
    int * A, * B, * C;

    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];

    initialize(A, vectorSize);
    initialize(B, vectorSize);

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    int * X, * Y, * Z;
    
    //memory for the vectors is allocated on the GPU using cudaMalloc.
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);
    
    //The cudaMemcpy function is used to copy data from the host vectors CPU (A and B) to the device vectors GPU(X and Y).
    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    //The kernel launch parameters are defined: threadsPerBlock is set to 256 threads per block for good balance between performance and efficiency, 
    //and blocksPerGrid is calculated to ensure all elements of the vectors are processed.
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    //The add function adds elements from vectors X and Y and stores the result in vector Z.
    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);
    
    //The result vector Z on the device is copied back to the host using cudaMemcpy.
    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

    cout << "Addition: ";
    print(C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;

    // nvcc 4_vecadd.cu -o 4_vecadd && ./4_vecadd
}

/*
CUDA (Compute Unified Device Architecture) is a parallel computing platform and programming model developed by NVIDIA for general-purpose computing on its graphics processing units (GPUs). It allows developers to write programs that can execute concurrently on thousands of small processing cores within a GPU, making it ideal for accelerating complex computations, such as scientific simulations, image processing, and machine learning. CUDA provides an API and language extensions (based on C/C++) for creating and managing GPU kernels, memory, and synchronization.
*/
